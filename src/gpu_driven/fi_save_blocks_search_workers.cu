#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string.h>
#include <math.h>
#include <stdlib.h>

extern "C" {
#include "../image_misc.h"
#include "../fi_save_blocks_search_workers.h"
}

#define IMAGE_MAXCUDABLOCKSIZE 64

static void ShowMeSomeCUDAInfo(void);

typedef struct {
	unsigned int dx; // Смещение по x доменного блока
	unsigned int dy; // Смещение по у доменного блока
	unsigned int tr; // Трансформация
	unsigned int u; // Контрастность
	int v; // Яркость
	unsigned int diff;
} kernel_block_type;

__device__ void fiCalcReversedTransformationNone(long long &range_sum, long long &domain_sum, long long &rd_sum, long long &domain_disp, int *range_block,
	unsigned int dx, unsigned int dy,
	unsigned int rx, unsigned int ry, unsigned int blocksize,
	unsigned int w, unsigned int h,
	int *scaled_data_cuda,
	int *data_cuda
	)
{
	unsigned int l, m;

	rd_sum = domain_disp = domain_sum = range_sum = 0;

	for(l = 0; l < blocksize; l++) {
		for(m = 0; m < blocksize; m++) {
			int br = data_cuda[(ry+l)*w+rx+m];
			int bd = scaled_data_cuda[(dy+l)*w/2+dx+m];

			range_block[l*blocksize+m] = br;

			range_sum += range_block[l*blocksize+m];

			domain_sum += bd;
			domain_disp += bd*bd;
			rd_sum += br*bd;
		}
	}

	domain_disp = domain_disp*(int)(blocksize*blocksize)-domain_sum*domain_sum;
}

__device__ void fiCalcReversedTransformation90Right(long long &range_sum, long long &domain_sum, long long &rd_sum, long long &domain_disp, int *range_block,
	unsigned int dx, unsigned int dy,
	unsigned int rx, unsigned int ry, unsigned int blocksize,
	unsigned int w, unsigned int h,
	int *scaled_data_cuda,
	int *data_cuda
	)
{
	unsigned int l, m;

	rd_sum = domain_disp = domain_sum = range_sum = 0;

	for(l = 0; l < blocksize; l++) {
		for(m = 0; m < blocksize; m++) {
			int br = data_cuda[(ry+m)*w+rx+blocksize-1-l];
			int bd = scaled_data_cuda[(dy+l)*w/2+dx+m];

			range_block[l*blocksize+m] = br;

			range_sum += range_block[l*blocksize+m];

			domain_sum += bd;
			domain_disp += bd*bd;
			rd_sum += br*bd;
		}
	}

	domain_disp = domain_disp*(int)(blocksize*blocksize)-domain_sum*domain_sum;
}

__device__ void fiCalcReversedTransformation180Right(long long &range_sum, long long &domain_sum, long long &rd_sum, long long &domain_disp, int *range_block,
	unsigned int dx, unsigned int dy,
	unsigned int rx, unsigned int ry, unsigned int blocksize,
	unsigned int w, unsigned int h,
	int *scaled_data_cuda,
	int *data_cuda
	)
{
	unsigned int l, m;

	rd_sum = domain_disp = domain_sum = range_sum = 0;

	for(l = 0; l < blocksize; l++) {
		for(m = 0; m < blocksize; m++) {
			int br = data_cuda[(ry+blocksize-1-l)*w+rx+blocksize-1-m];
			int bd = scaled_data_cuda[(dy+l)*w/2+dx+m];

			range_block[l*blocksize+m] = br;

			range_sum += range_block[l*blocksize+m];

			domain_sum += bd;
			domain_disp += bd*bd;
			rd_sum += br*bd;
		}
	}

	domain_disp = domain_disp*(int)(blocksize*blocksize)-domain_sum*domain_sum;
}

__device__ void fiCalcReversedTransformation270Right(long long &range_sum, long long &domain_sum, long long &rd_sum, long long &domain_disp, int *range_block,
	unsigned int dx, unsigned int dy,
	unsigned int rx, unsigned int ry, unsigned int blocksize,
	unsigned int w, unsigned int h,
	int *scaled_data_cuda,
	int *data_cuda
	)
{
	unsigned int l, m;

	rd_sum = domain_disp = domain_sum = range_sum = 0;

	for(l = 0; l < blocksize; l++) {
		for(m = 0; m < blocksize; m++) {
			int br = data_cuda[(ry+blocksize-1-m)*w+rx+l];
			int bd = scaled_data_cuda[(dy+l)*w/2+dx+m];

			range_block[l*blocksize+m] = br;

			range_sum += range_block[l*blocksize+m];

			domain_sum += bd;
			domain_disp += bd*bd;
			rd_sum += br*bd;
		}
	}

	domain_disp = domain_disp*(int)(blocksize*blocksize)-domain_sum*domain_sum;
}

__device__ void fiCalcReversedTransformationVFlip(long long &range_sum, long long &domain_sum, long long &rd_sum, long long &domain_disp, int *range_block,
	unsigned int dx, unsigned int dy,
	unsigned int rx, unsigned int ry, unsigned int blocksize,
	unsigned int w, unsigned int h,
	int *scaled_data_cuda,
	int *data_cuda
	)
{
	unsigned int l, m;

	rd_sum = domain_disp = domain_sum = range_sum = 0;

	for(l = 0; l < blocksize; l++) {
		for(m = 0; m < blocksize; m++) {
			int br = data_cuda[(ry+blocksize-1-l)*w+rx+m];
			int bd = scaled_data_cuda[(dy+l)*w/2+dx+m];

			range_block[l*blocksize+m] = br;

			range_sum += range_block[l*blocksize+m];

			domain_sum += bd;
			domain_disp += bd*bd;
			rd_sum += br*bd;
		}
	}

	domain_disp = domain_disp*(int)(blocksize*blocksize)-domain_sum*domain_sum;
}

__device__ void fiCalcReversedTransformationHFlip(long long &range_sum, long long &domain_sum, long long &rd_sum, long long &domain_disp, int *range_block,
	unsigned int dx, unsigned int dy,
	unsigned int rx, unsigned int ry, unsigned int blocksize,
	unsigned int w, unsigned int h,
	int *scaled_data_cuda,
	int *data_cuda
	)
{
	unsigned int l, m;

	rd_sum = domain_disp = domain_sum = range_sum = 0;

	for(l = 0; l < blocksize; l++) {
		for(m = 0; m < blocksize; m++) {
			int br = data_cuda[(ry+l)*w+rx+blocksize-1-m];
			int bd = scaled_data_cuda[(dy+l)*w/2+dx+m];

			range_block[l*blocksize+m] = br;

			range_sum += range_block[l*blocksize+m];

			domain_sum += bd;
			domain_disp += bd*bd;
			rd_sum += br*bd;
		}
	}

	domain_disp = domain_disp*(int)(blocksize*blocksize)-domain_sum*domain_sum;
}

__device__ void fiCalcReversedTransformationMainDiagFlip(long long &range_sum, long long &domain_sum, long long &rd_sum, long long &domain_disp, int *range_block,
	unsigned int dx, unsigned int dy,
	unsigned int rx, unsigned int ry, unsigned int blocksize,
	unsigned int w, unsigned int h,
	int *scaled_data_cuda,
	int *data_cuda
	)
{
	unsigned int l, m;

	rd_sum = domain_disp = domain_sum = range_sum = 0;

	for(l = 0; l < blocksize; l++) {
		for(m = 0; m < blocksize; m++) {
			int br = data_cuda[(ry+m)*w+rx+l];
			int bd = scaled_data_cuda[(dy+l)*w/2+dx+m];

			range_block[l*blocksize+m] = br;

			range_sum += range_block[l*blocksize+m];

			domain_sum += bd;
			domain_disp += bd*bd;
			rd_sum += br*bd;
		}
	}

	domain_disp = domain_disp*(int)(blocksize*blocksize)-domain_sum*domain_sum;
}

__device__ void fiCalcReversedTransformationAntiDiagFlip(long long &range_sum, long long &domain_sum, long long &rd_sum, long long &domain_disp, int *range_block,
	unsigned int dx, unsigned int dy,
	unsigned int rx, unsigned int ry, unsigned int blocksize,
	unsigned int w, unsigned int h,
	int *scaled_data_cuda,
	int *data_cuda
	)
{
	unsigned int l, m;

	rd_sum = domain_disp = domain_sum = range_sum = 0;

	for(l = 0; l < blocksize; l++) {
		for(m = 0; m < blocksize; m++) {
			int br = data_cuda[(ry+blocksize-1-m)*w+rx+blocksize-1-l];
			int bd = scaled_data_cuda[(dy+l)*w/2+dx+m];

			range_block[l*blocksize+m] = br;

			range_sum += range_block[l*blocksize+m];

			domain_sum += bd;
			domain_disp += bd*bd;
			rd_sum += br*bd;
		}
	}

	domain_disp = domain_disp*(int)(blocksize*blocksize)-domain_sum*domain_sum;
}

__global__ void fiFindBestDomainBlockKernel(unsigned int rx, unsigned int ry,
	kernel_block_type *kernel_blocks_cuda, unsigned int blocksize,
	unsigned int w, unsigned int h,
	int *scaled_data_cuda,
	int *data_cuda
	)
{
	/*__shared__ */int range_block[IMAGE_MAXCUDABLOCKSIZE*IMAGE_MAXCUDABLOCKSIZE];
	long long range_sum, domain_sum, rd_sum; // Сумма пикселей рангового блока, доменного блока, пикселя рангового на пиксель доменного блока
	long long domain_disp;
	unsigned int diff;
	int u; int v;
	unsigned int mono_diff;
	int mono_u; int mono_v;
	unsigned int local_diff;
	__shared__ unsigned int best_diff;
	int local_u; int local_v;
	unsigned int dx, dy; // Координаты доменного блока
	unsigned int tr; // Трансформация, вычисляемая в текущем потоке
	kernel_block_type *kernel_block_cuda; // Текущий доменный блок для текущего блока cuda

	unsigned int l, m;

	atomicExch(&best_diff, 255*255*blocksize*blocksize*(blockDim.x*blockDim.y)+(blockDim.x*blockDim.y));
	__syncthreads();

	kernel_block_cuda = kernel_blocks_cuda+blockIdx.z*gridDim.y*gridDim.x+blockIdx.y*gridDim.x+blockIdx.x;

	dx = (blockIdx.x*blockDim.x+threadIdx.x)%(w/2-blocksize+1);
	dy = (blockIdx.y*blockDim.y+threadIdx.y)%(h/2-blocksize+1);
	tr = blockIdx.z+threadIdx.z;

	switch(tr) {
		case IMAGE_RB_TRANSFORMATION_NONE:
			fiCalcReversedTransformationNone(range_sum, domain_sum, rd_sum, domain_disp, range_block,
				dx, dy,
				rx, ry, blocksize,
				w, h,
				scaled_data_cuda,
				data_cuda
				);
			break;
		case IMAGE_RB_TRANSFORMATION_90RIGHT:
			fiCalcReversedTransformation90Right(range_sum, domain_sum, rd_sum, domain_disp, range_block,
				dx, dy,
				rx, ry, blocksize,
				w, h,
				scaled_data_cuda,
				data_cuda
				);
			break;
		case IMAGE_RB_TRANSFORMATION_180RIGHT:
			fiCalcReversedTransformation180Right(range_sum, domain_sum, rd_sum, domain_disp, range_block,
				dx, dy,
				rx, ry, blocksize,
				w, h,
				scaled_data_cuda,
				data_cuda
				);
			break;
		case IMAGE_RB_TRANSFORMATION_270RIGHT:
			fiCalcReversedTransformation270Right(range_sum, domain_sum, rd_sum, domain_disp, range_block,
				dx, dy,
				rx, ry, blocksize,
				w, h,
				scaled_data_cuda,
				data_cuda
				);
			break;
		case IMAGE_RB_TRANSFORMATION_VFLIP:
			fiCalcReversedTransformationVFlip(range_sum, domain_sum, rd_sum, domain_disp, range_block,
				dx, dy,
				rx, ry, blocksize,
				w, h,
				scaled_data_cuda,
				data_cuda
				);
			break;
		case IMAGE_RB_TRANSFORMATION_HFLIP:
			fiCalcReversedTransformationHFlip(range_sum, domain_sum, rd_sum, domain_disp, range_block,
				dx, dy,
				rx, ry, blocksize,
				w, h,
				scaled_data_cuda,
				data_cuda
				);
			break;
		case IMAGE_RB_TRANSFORMATION_MAINDIAGFLIP:
			fiCalcReversedTransformationMainDiagFlip(range_sum, domain_sum, rd_sum, domain_disp, range_block,
				dx, dy,
				rx, ry, blocksize,
				w, h,
				scaled_data_cuda,
				data_cuda
				);
			break;
		case IMAGE_RB_TRANSFORMATION_ANTIDIAGFLIP:
			fiCalcReversedTransformationAntiDiagFlip(range_sum, domain_sum, rd_sum, domain_disp, range_block,
				dx, dy,
				rx, ry, blocksize,
				w, h,
				scaled_data_cuda,
				data_cuda
				);
			break;
	}

	mono_u = 0; mono_v = (int)( ((range_sum/(blocksize*blocksize))*63+128)/255 ); // Значения на случай, если совсем ни один блок не подойдёт

	if(domain_disp) {
		u = (int)( 32*((int)(blocksize*blocksize)*rd_sum-range_sum*domain_sum)/domain_disp ); // яркость в интервале [0, 1.0), т.е. от [0 до 32)
		if(u < 0) u = 0; if(u > 31) u = 31;
		v = (int)( (range_sum-domain_sum*u/32)/(int)(blocksize*blocksize) );
		if(v < -255 || v > 255) {
			u = mono_u;
			v = mono_v;
		} else {
			if(v < 0)
				v = (v*63-128)/255;
			else
				v = (v*63+128)/255;
		}
	} else {
		u = mono_u;
		v = mono_v;
	}

	mono_diff = 0; // Найдём разницу для mono_v и mono_u (mono_diff)
	diff = 0; // Найдём разницу для v и u блока dx, dy (diff)
	for(l = 0; l < blocksize; l++) {
		for(m = 0; m < blocksize; m++) {
			int br = range_block[l*blocksize+m];
			int bd = scaled_data_cuda[(dy+l)*w/2+dx+m];

			int mult;

			mult = (mono_v*255/63-br);
			mono_diff += mult*mult;

			mult = (bd*u/32+v*255/63-br);
			diff += mult*mult;
		}
	}

	if(diff < mono_diff) {
		local_diff = diff;
		local_u = u;
		local_v = v;
	} else {
		local_diff = mono_diff;
		local_u = mono_u;
		local_v = mono_v;
	}

	unsigned int uniq_diff = (unsigned int)sqrtf(local_diff)*(blockDim.x*blockDim.y)+(threadIdx.y*blockDim.x+threadIdx.x); // 255*(IMAGE_MAXCUDABLOCKSIZE=64)*64*64=3FC000 < FFFFFFFF
	atomicMin(&best_diff, uniq_diff);
	__syncthreads();

	if(best_diff == uniq_diff) {
		kernel_block_cuda->dx = dx;
		kernel_block_cuda->dy = dy;
		kernel_block_cuda->tr = tr;
		kernel_block_cuda->u = local_u;
		kernel_block_cuda->v = local_v;
		kernel_block_cuda->diff = local_diff;
	}
}

static void fiFindBestDomainBlock(unsigned int rx, unsigned int ry,
	image_rangeblock_type *block_p, kernel_block_type *kernel_blocks_cuda, kernel_block_type *kernel_blocks, unsigned int blocksize,
	unsigned int w, unsigned int h,
	dim3 threads, dim3 blocks,
	int *scaled_data_cuda,
	int *data_cuda,
	unsigned int worst_diff
	)
{
	unsigned int best_diff, i;
	kernel_block_type *kernel_block;

	memset(kernel_blocks, 0, sizeof(kernel_block_type)*blocks.x*blocks.y*blocks.z);
	hipMemcpy(kernel_blocks_cuda, kernel_blocks, sizeof(kernel_block_type)*blocks.x*blocks.y*blocks.z, hipMemcpyHostToDevice);

	fiFindBestDomainBlockKernel <<<blocks, threads>>>(rx, ry, kernel_blocks_cuda, blocksize, w, h, scaled_data_cuda, data_cuda);

	hipMemcpy(kernel_blocks, kernel_blocks_cuda, sizeof(kernel_block_type)*blocks.x*blocks.y*blocks.z, hipMemcpyDeviceToHost);

	best_diff = kernel_blocks->diff;
	kernel_block = kernel_blocks;
	for(i = 0; i < blocks.x*blocks.y*blocks.z; i++) {
		if(kernel_blocks[i].diff < best_diff) {
			best_diff = kernel_blocks[i].diff;
			kernel_block = kernel_blocks+i;

		}
		//printf("-? rx %u ry %u i %u dx %u dy %u tr %u l_u %u l_v %d l_diff %u\n", rx, ry, i, kernel_blocks[i].dx, kernel_blocks[i].dy, kernel_blocks[i].tr, kernel_blocks[i].u, kernel_blocks[i].v, kernel_blocks[i].diff);
	}

	block_p->dx = kernel_block->dx;
	block_p->dy = kernel_block->dy;
	block_p->u = kernel_block->u;
	block_p->v = kernel_block->v;
	block_p->tr = kernel_block->tr;
	//printf("-! rx %u ry %u dx %u dy %u tr %u l_u %u l_v %d l_diff %u\n", rx, ry, kernel_block->dx, kernel_block->dy, kernel_block->tr, kernel_block->u, kernel_block->v, kernel_block->diff);

	if(best_diff > worst_diff && blocksize > 2 && blocksize%2 == 0) {
		printf("best_diff %d/%d (=rms^2*blocksize^2), divided block size %d to %d\n", best_diff, worst_diff, blocksize, blocksize/2);

		fiFindBestDomainBlock(rx, ry, // Позиция нового блока блока
			(image_rangeblock_type *)block_p->divided_into, kernel_blocks_cuda, kernel_blocks, // Указатель на новый блок, на блоки для вычислений на cuda, память под них на cpu
			blocksize/2, // Уменьшаем размер блока на 2
			w, h, threads, blocks, scaled_data_cuda, data_cuda,
			worst_diff/4); // Уменьшаем разницу на 4 (т.к. новый блок будет в 4 раза меньше)

		fiFindBestDomainBlock(rx+blocksize/2, ry, // Позиция нового блока блока
			(image_rangeblock_type *)block_p->divided_into+1, kernel_blocks_cuda, kernel_blocks, // Указатель на новый блок, на блоки для вычислений на cuda, память под них на cpu
			blocksize/2, // Уменьшаем размер блока на 2
			w, h, threads, blocks, scaled_data_cuda, data_cuda,
			worst_diff/4); // Уменьшаем разницу на 4 (т.к. новый блок будет в 4 раза меньше)

		fiFindBestDomainBlock(rx, ry+blocksize/2, // Позиция нового блока блока
			(image_rangeblock_type *)block_p->divided_into+2, kernel_blocks_cuda, kernel_blocks, // Указатель на новый блок, на блоки для вычислений на cuda, память под них на cpu
			blocksize/2, // Уменьшаем размер блока на 2
			w, h, threads, blocks, scaled_data_cuda, data_cuda,
			worst_diff/4); // Уменьшаем разницу на 4 (т.к. новый блок будет в 4 раза меньше)

		fiFindBestDomainBlock(rx+blocksize/2, ry+blocksize/2, // Позиция нового блока блока
			(image_rangeblock_type *)block_p->divided_into+3, kernel_blocks_cuda, kernel_blocks, // Указатель на новый блок, на блоки для вычислений на cuda, память под них на cpu
			blocksize/2, // Уменьшаем размер блока на 2
			w, h, threads, blocks, scaled_data_cuda, data_cuda,
			worst_diff/4); // Уменьшаем разницу на 4 (т.к. новый блок будет в 4 раза меньше)

		block_p->is_divided = true;
	}
}

#ifdef WIN32
unsigned int __stdcall fiWorkerFunc(void *arg)
#else
void *fiWorkerFunc(void *arg)
#endif
{
	unsigned int i;
	unsigned int rx, ry, channel, copied_channel;

	unsigned char *scaled_data = ((fi_worker_arg_type *)arg)->scaled_data;
	unsigned char **data = ((fi_worker_arg_type *)arg)->data;
	int *data_in_int = 0;
	image_rangeblock_type *block_start_p = ((fi_worker_arg_type *)arg)->block_p, *block_p;
	unsigned int nof_blocks_per_channel = ((fi_worker_arg_type *)arg)->nof_blocks_per_channel;
	unsigned int w = ((fi_worker_arg_type *)arg)->w;
	unsigned int h = ((fi_worker_arg_type *)arg)->h;
	unsigned int nof_channels = ((fi_worker_arg_type *)arg)->nof_channels;
	unsigned int alloc_memory = ((fi_worker_arg_type *)arg)->alloc_memory;
	unsigned int blocksize = ((fi_worker_arg_type *)arg)->blocksize;
	unsigned int worker_start = ((fi_worker_arg_type *)arg)->worker_start;
	unsigned int worker_step = ((fi_worker_arg_type *)arg)->worker_step;
	unsigned int worst_diff = ((fi_worker_arg_type *)arg)->worst_diff;

	unsigned int nof_blocks_per_image = nof_blocks_per_channel*nof_channels;

	int *scaled_data_cuda;
	kernel_block_type *kernel_blocks_cuda;
	kernel_block_type *kernel_blocks;
	int *data_cuda;

	hipEvent_t start, stop;
	double cuda_working_time = 0;

	hipDeviceProp_t device_prop;
	dim3 threads, blocks;

	ShowMeSomeCUDAInfo();

	if(blocksize > IMAGE_MAXCUDABLOCKSIZE) {
		printf("Sorry, but IMAGE_MAXCUDABLOCKSIZE=%d < blocksize=%d\b", IMAGE_MAXCUDABLOCKSIZE, blocksize);

		return 0;
	}

	printf("hipMalloc(&scaled_data_cuda, alloc_memory/4=%d)\n", sizeof(int)*alloc_memory/4);
	if(hipMalloc(&scaled_data_cuda, sizeof(int)*alloc_memory/4) != hipSuccess) {
		printf("Can\'t allocate memory for scaled_data_cuda\n");

		return 0;
	}
	printf("hipMalloc(&data_cuda, alloc_memory=%d)\n", sizeof(int)*alloc_memory);
	if(hipMalloc(&data_cuda, sizeof(int)*alloc_memory) != hipSuccess) {
		printf("Can\'t allocate memory for data_cuda_p\n");

		hipFree(scaled_data_cuda);

		return 0;
	}

	if(hipEventCreate(&start) != hipSuccess) {
		printf("Can\'t create event \"start\"\n");

		hipFree(scaled_data_cuda);
		hipFree(data_cuda);

		return 0;
	}
	if(hipEventCreate(&stop) != hipSuccess) {
		printf("Can\'t create event \"stop\"\n");

		hipFree(scaled_data_cuda);
		hipFree(data_cuda);

		hipEventDestroy(start);

		return 0;
	}

	hipGetDeviceProperties(&device_prop, 0);

	int number_of_working_threads = device_prop.maxThreadsPerBlock/2; // Магия
	int number_of_working_threads_multiplier = 1;
	int threads_z = 1;

	if(device_prop.maxGridSize[2] < 8) {
		number_of_working_threads /= 8;
		threads_z = 8;
	}

	while( ((unsigned int)(sqrt((float)number_of_working_threads)+0.5)*(unsigned int)(sqrt((float)number_of_working_threads)+0.5)) != number_of_working_threads) {
		number_of_working_threads /= 2;
		number_of_working_threads_multiplier *= 2;
	}

	threads = dim3((unsigned int)(sqrt((float)number_of_working_threads)+0.5)*number_of_working_threads_multiplier, (unsigned int)(sqrt((float)number_of_working_threads)+0.5), threads_z);
	blocks = dim3(((w/2-blocksize+1)+threads.x-1)/threads.x, ((h/2-blocksize+1)+threads.y-1)/threads.y, 8/threads.z);

	printf("threads(%u,%u,%u) blocks(%u,%u,%u)\n", threads.x, threads.y, threads.z, blocks.x, blocks.y, blocks.z);

	if((int)(threads.x*threads.y*threads.z) > device_prop.maxThreadsPerBlock) {
		printf("threads.x*threads.y*threads.z is greater than %d, exiting...\n", device_prop.maxThreadsPerBlock);

		hipFree(scaled_data_cuda);
		hipFree(data_cuda);

		hipEventDestroy(start);
		hipEventDestroy(stop);

		return 0;
	}

	printf("hipMalloc(&kernel_blocks_cuda, sizeof(image_rangeblock_type)*blocks.x*blocks.y*blocks.z=%d)\n", sizeof(kernel_block_type)*blocks.x*blocks.y*blocks.z);
	if(hipMalloc(&kernel_blocks_cuda, sizeof(kernel_block_type)*blocks.x*blocks.y*blocks.z) != hipSuccess) {
		printf("Can\'t allocate memory for block_start_cuda_p\n");

		hipFree(scaled_data_cuda);
		hipFree(data_cuda);

		hipEventDestroy(start);
		hipEventDestroy(stop);

		return 0;
	}

	printf("kernel_blocks = malloc(sizeof(kernel_block_type)*blocks.x*blocks.y*blocks.z=%d), sizeof(kernel_block_type)=%d\n", sizeof(kernel_block_type)*blocks.x*blocks.y*blocks.z, sizeof(kernel_block_type));
	kernel_blocks = (kernel_block_type *)malloc(sizeof(kernel_block_type)*blocks.x*blocks.y*blocks.z);
	if(kernel_blocks == 0) {
		printf("Can\'t allocate memory for block_start_cuda_p\n");

		hipFree(scaled_data_cuda);
		hipFree(data_cuda);
		hipFree(kernel_blocks_cuda);

		hipEventDestroy(start);
		hipEventDestroy(stop);

		return 0;
	}

	copied_channel = nof_channels+1;
	block_p = block_start_p+worker_start;

	for(i = worker_start; i < nof_blocks_per_image; i += worker_step) {
		float one_block_time;

		channel = i / nof_blocks_per_channel;
		rx = ((i % nof_blocks_per_channel) % (w/blocksize)) * blocksize;
		ry = ((i % nof_blocks_per_channel) / (w/blocksize)) * blocksize;

		if(copied_channel != channel) {
			unsigned int j;

			if(!data_in_int)
				data_in_int = (int *)malloc(sizeof(int)*alloc_memory);

			if(!data_in_int) {
				printf("Can\'t allocate memory for data_in_int\n");

				break;
			}

			copied_channel = channel;

			for(j = 0; j < alloc_memory/4; j++) {
				data_in_int[j] = scaled_data[channel*alloc_memory/4+j];
			}
			hipMemcpy(scaled_data_cuda, data_in_int, sizeof(int)*alloc_memory/4, hipMemcpyHostToDevice);

			for(j = 0; j < alloc_memory; j++) {
				data_in_int[j] = data[channel][j];
			}
			hipMemcpy(data_cuda, data_in_int, sizeof(int)*alloc_memory, hipMemcpyHostToDevice);
		}

		if(((i-worker_start)/worker_step)%10 == 0)
			printf("worker %d %d rbs (%f) ok\n", worker_start+1, (i-worker_start)/worker_step,
				(float)( 100.0 * (double)((i-worker_start)/worker_step) / (double)((nof_blocks_per_image-worker_start-1)/worker_step+1) ));

		hipEventRecord(start, 0);

		fiFindBestDomainBlock(rx, ry,
			block_p, kernel_blocks_cuda, kernel_blocks, blocksize,
			w, h,
			threads, blocks,
			scaled_data_cuda,
			data_cuda,
			worst_diff);

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&one_block_time, start, stop);
		cuda_working_time += one_block_time;

		block_p += worker_step;
	}

	if(data_in_int)
		free(data_in_int);
	hipFree(scaled_data_cuda);
	hipFree(data_cuda);
	hipFree(kernel_blocks_cuda);
	free(kernel_blocks);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	if(worker_step == 1) { // Сбрасываем устройство, только если имеется только один поток
		if(hipDeviceReset() != hipSuccess) {
			printf("Can\'t reset cuda device\n");
		}
	}

	printf("cuda_working_time %f ms\n", cuda_working_time);

	return 0;
}

static void ShowMeSomeCUDAInfo(void)
{
	int device_count;
	hipDeviceProp_t device_prop;

	hipGetDeviceCount(&device_count);

	printf("Device count: %d\n\n", device_count);

	for(int i = 0; i < device_count; i++) {
		hipGetDeviceProperties(&device_prop, i);

		printf("Device name: %s\n", device_prop.name);
		printf("Compute capability: %d.%d\n", device_prop.major, device_prop.minor);
		printf("Total constant memory: %lld\n", (long long)(device_prop.totalConstMem));
		printf("Total global memory: %lld\n", (long long)(device_prop.totalGlobalMem));
		printf("Shared memory per block: %lld\n", (long long)(device_prop.sharedMemPerBlock));
		printf("Registers per block: %d\n", device_prop.regsPerBlock);
		printf("Warp size: %d\n", device_prop.warpSize);
		printf("Memory pitch: %lld\n", (long long)(device_prop.memPitch));
		printf("Max threads per block: %d\n", device_prop.maxThreadsPerBlock);
		printf("Max threads dimensions: x = %d, y = %d, z = %d\n",
			device_prop.maxThreadsDim[0],
			device_prop.maxThreadsDim[1],
			device_prop.maxThreadsDim[2]);

		printf("Max grid size: x = %d, y = %d, z = %d\n",
			device_prop.maxGridSize[0],
			device_prop.maxGridSize[1],
			device_prop.maxGridSize[2]);

		printf("Max Surface1D: %d\n", device_prop.maxSurface1D);
		printf("Max Surface2D: %d\n", device_prop.maxSurface2D);

		printf("Clock rate: %d\n", device_prop.clockRate);
		printf("Total constant memory: %d\n", device_prop.totalConstMem);
		printf("Compute capability: %d.%d\n", device_prop.major, device_prop.minor);
		printf("Texture alignment: %d\n", device_prop.textureAlignment);
		printf("Device overlap: %d\n", device_prop.deviceOverlap);
		printf("Multiprocessor count: %d\n", device_prop.multiProcessorCount);
		printf("Kernel execution timeout enabled: %s\n",
			device_prop.kernelExecTimeoutEnabled ? "true" : "false");
	}

	printf("\n");

}
